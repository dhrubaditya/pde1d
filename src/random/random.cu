#include "random.h"
#include <iostream>

static hiprandGenerator_t gen;
static bool initialized = false;

__host__ void rng_init(unsigned long long seed)
{
    if (initialized) return;

    hiprandStatus_t status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
    if (status != HIPRAND_STATUS_SUCCESS) {
        std::cerr << "Error: Failed to create cuRAND MTGP32 generator\n";
        return;
    }

    status = hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    if (status != HIPRAND_STATUS_SUCCESS) {
        std::cerr << "Error: Failed to set cuRAND seed\n";
        return;
    }

    initialized = true;
}

__host__ void rng_generate_uniform(double *d_data, size_t N)
{
    if (!initialized) {
        std::cerr << "Error: RNG not initialized. Call rng_init() first.\n";
        return;
    }

    hiprandStatus_t status = hiprandGenerateUniformDouble(gen, d_data, N);
    if (status != HIPRAND_STATUS_SUCCESS) {
        std::cerr << "Error: cuRAND failed to generate uniform doubles\n";
    }
}

__host__ void rng_generate_normal(double *d_data, size_t N, double mean, double stddev)
{
    if (!initialized) {
        std::cerr << "Error: RNG not initialized. Call rng_init() first.\n";
        return;
    }

    hiprandStatus_t status = hiprandGenerateNormalDouble(gen, d_data, N, mean, stddev);
    if (status != HIPRAND_STATUS_SUCCESS) {
        std::cerr << "Error: cuRAND failed to generate normal doubles\n";
    }
}

__host__ void rng_destroy()
{
    if (!initialized) return;
    hiprandDestroyGenerator(gen);
    initialized = false;
}

