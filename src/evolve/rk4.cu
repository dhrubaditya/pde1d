#include "hip/hip_runtime.h"
// rk4.cu
#include "rk4.h"
#include "harmonic.h"   // declaration of rhs_kernel and energy kernel wrapper
#include <cstdio>

#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = (call);                                           \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "CUDA error %s:%d: '%s'\n", __FILE__, __LINE__, \
                    hipGetErrorString(err));                               \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    } while (0)

static const int BLOCK_SIZE = RK4_BLOCK_SIZE;

// ---------------- kernels ----------------
// combine stage: Ytemp = Y + a * k
__global__ void combine_stage_kernel(double* Ytemp,
                                     const double* Y,
                                     const double* k,
                                     double a,
                                     int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) Ytemp[i] = Y[i] + a * k[i];
}

// final RK4 update: Y <- Y + (k1 + 2*k2 + 2*k3 + k4)/6
__global__ void rk4_update_kernel(double* Y,
                                  const double* k1,
                                  const double* k2,
                                  const double* k3,
                                  const double* k4,
                                  int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        Y[i] += (k1[i] + 2.0 * k2[i] + 2.0 * k3[i] + k4[i]) / 6.0;
    }
}

// ---------------- host functions ----------------
RK4DeviceData rk4_allocate_device_memory(int N)
{
    RK4DeviceData dev;
    dev.is_initialized = false;

    CUDA_CHECK(hipMalloc(&dev.d_Y,     N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dev.d_Ytemp, N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dev.d_k1,    N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dev.d_k2,    N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dev.d_k3,    N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&dev.d_k4,    N * sizeof(double)));

    return dev;
}

void rk4_free_device_memory(RK4DeviceData& dev)
{
    if (dev.d_Y)     hipFree(dev.d_Y);
    if (dev.d_Ytemp) hipFree(dev.d_Ytemp);
    if (dev.d_k1)    hipFree(dev.d_k1);
    if (dev.d_k2)    hipFree(dev.d_k2);
    if (dev.d_k3)    hipFree(dev.d_k3);
    if (dev.d_k4)    hipFree(dev.d_k4);

    dev.d_Y = dev.d_Ytemp = dev.d_k1 = dev.d_k2 = dev.d_k3 = dev.d_k4 = nullptr;
    dev.is_initialized = false;
}

// Perform M RK4 timesteps. The RHS for each stage is computed by a single
// device-parallel kernel rhs_kernel which writes dt*f(Y) into the k arrays.
void rk4_timestep_host(double* Y_host, int N, double dt, int M, RK4DeviceData& dev)
{
    if (N <= 0) return;
    const int threads = BLOCK_SIZE;
    const int blocks  = (N + threads - 1) / threads;

    // Copy host->device only on first call
    if (!dev.is_initialized) {
        CUDA_CHECK(hipMemcpy(dev.d_Y, Y_host, N * sizeof(double), hipMemcpyHostToDevice));
        dev.is_initialized = true;
    }

    for (int step = 0; step < M; ++step) {
        // Stage 1: k1 = dt * f(Y) -> dev.d_k1
        rhs_kernel<<<blocks, threads>>>(dev.d_Y, dev.d_k1, dt, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Stage 2: Ytemp = Y + 0.5*k1 ; k2 = dt * f(Ytemp)
        combine_stage_kernel<<<blocks, threads>>>(dev.d_Ytemp, dev.d_Y, dev.d_k1, 0.5, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        rhs_kernel<<<blocks, threads>>>(dev.d_Ytemp, dev.d_k2, dt, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Stage 3: Ytemp = Y + 0.5*k2 ; k3 = dt * f(Ytemp)
        combine_stage_kernel<<<blocks, threads>>>(dev.d_Ytemp, dev.d_Y, dev.d_k2, 0.5, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        rhs_kernel<<<blocks, threads>>>(dev.d_Ytemp, dev.d_k3, dt, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Stage 4: Ytemp = Y + k3 ; k4 = dt * f(Ytemp)
        combine_stage_kernel<<<blocks, threads>>>(dev.d_Ytemp, dev.d_Y, dev.d_k3, 1.0, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        rhs_kernel<<<blocks, threads>>>(dev.d_Ytemp, dev.d_k4, dt, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Final update Y <- Y + (k1 + 2*k2 + 2*k3 + k4) / 6
        rk4_update_kernel<<<blocks, threads>>>(dev.d_Y, dev.d_k1, dev.d_k2, dev.d_k3, dev.d_k4, N);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }

    // Copy updated solution back to host
    CUDA_CHECK(hipMemcpy(Y_host, dev.d_Y, N * sizeof(double), hipMemcpyDeviceToHost));
}

