#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include "fft_utils.h"
#include "misc.h"
#include "random.h"
// Kernel to initialize real-space array: f(x) = sin(5x)
__global__ void init_sin_kernel(double* data, int N, double L) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        double dx = L / N;
        double x = i * dx;
        data[i] = sin(50.0 * x);
    }
}
// Kernel to set complex array
__global__ void complexify(hipfftDoubleComplex* Z, double* re, double* im, int N)
{	
    int nfreqs = N / 2 + 1;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nfreqs) {
      Z[i].x = re[i];
      Z[i].y = im[i];
    }
}
// Kernel to set complex array
__global__ void set_complex(hipfftDoubleComplex* Z, int N)
{	
    int nfreqs = N / 2 + 1;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nfreqs) {
      Z[i].x = 1.;
      Z[i].y = 0.;
      if (i == nfreqs -1){Z[i].y = 0.;}
      if (i == 0){Z[i].y = 0.;}
    }
}

int main(int argc, char** argv) {
    int N = 1024;              // number of real samples
    double L = 2 * M_PI;       // domain size
    if (argc > 1) {
      N = std::atoi(argv[1]);
    }
    printf("Grid size N = %d\n", N);
    // ----------------------------
    // Allocate FFT array and plan
    // ----------------------------
    FFTArray1D arr = fft_alloc_1d(N);
    // ----------------------------
    // host allocation
    // ----------------------------
    /*hipfftDoubleComplex* f1;
    hipfftDoubleComplex* f2;
    hipHostMalloc( (void**)&f1, sizeof(hipfftDoubleComplex) * (N/2 + 1) );
    hipHostMalloc( (void**)&f2, sizeof(hipfftDoubleComplex) * (N/2 + 1) );*/
    int block = 256;
    int grid = (N + block - 1) / block;
    /*init_sin_kernel<<<grid, block>>>(arr.d_real, N, L);
    GpuReducer red;
    init_reducer(red, N);
    double sum = gpu_sum(arr.d_real, N, red);
    std:: cout << sum << "\n"  ;*/
    /*double* d_real;
    double* d_imag;
    hipMalloc(&d_real, sizeof(double) * (N/2 + 1));
    hipMalloc(&d_imag, sizeof(double) * (N/2 + 1));
    unsigned long long seed = static_cast<unsigned long long>(time(nullptr));
    rng_init(seed);
    rng_generate_uniform(d_real, N/2 + 1);
    rng_generate_uniform(d_imag, N/2 + 1);*/
    set_complex<<<grid, block>>>(arr.d_complex, N);
    hipDeviceSynchronize();
    GpuComplexReducer ws;
    init_Complex_reducer(ws, N);
    hipfftDoubleComplex csum = gpu_Complex_sum(arr.d_complex,
                                       N, ws);
    std:: cout << csum.x << " " << csum.y << "\n"  ;
    /*hipMemcpy(f1, arr.d_complex, 
	       sizeof(hipfftDoubleComplex) * (N/2 + 1), 
	       hipMemcpyDeviceToHost);
    hipfftDoubleComplex II;
    II.x = 0;
    II.y = 1;
    complex_mult_FFTArray(arr, II); 
    // ----------------------------
    // ----------------------------
    hipMemcpy(f2, arr.d_complex, 
		    sizeof(hipfftDoubleComplex) * (N/2 + 1), 
		    hipMemcpyDeviceToHost); */
    // ----------------------------
    // Write to file
    // ----------------------------
    /* std::ofstream fout("data.txt");
    for (int k = 0; k < N/2 + 1; ++k) {
      fout << f1[k].x << " " << f1[k].y << " " << f2[k].x << " "
	   << f2[k].y << "\n";
    }
    fout.close(); */
    // ----------------------------
    // Clean up
    // ----------------------------
    /*hipFree(d_real);
    hipFree(d_imag);
    hipHostFree(f1);
    hipHostFree(f2); */
    fft_free_1d(arr);
    free_Complex_reducer(ws);
    //free_reducer(red);
    std::cout << "output written to data.txt\n";
    return 0;
}
