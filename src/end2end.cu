#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>
#include <cstdio>
#include "start.h"
#include "io.h"
#include "fft_utils.h"
#include "initcond.h"
#include "misc.h"
#include "model.h"
#define CUDA_CHECK(call)						\
    do {                                                                    \
        hipError_t err = (call);                                           \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "CUDA error %s:%d: '%s'\n", __FILE__, __LINE__, \
                    hipGetErrorString(err));                               \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    } while (0)
// ******************************************** //
int main() {
  // section 1 : Input  Parameters
    const SParams h_Sparams = read_Sparams("./input/start.in");

    std::cout << "Simulation parameters:" << std::endl;
    std::cout << "NX = " << h_Sparams.NX << std::endl;
    std::cout << "DX = " << h_Sparams.DX << std::endl;
    std::cout << "DK = " << h_Sparams.DK << std::endl;

// endsection
// section 2 : initial condition 
    int N = h_Sparams.NX;
    double dk = h_Sparams.DK;
    double dx = h_Sparams.DX;

    // --- Host memory  ---
    hipfftDoubleReal *psi;
    hipfftDoubleComplex *psik;
    CUDA_CHECK(hipHostMalloc((void**)&psi,
			      sizeof(hipfftDoubleReal) * (N + 2) ));
    CUDA_CHECK(hipHostMalloc((void**)&psik,
                               sizeof(hipfftDoubleComplex) * (N/2 + 1) ));
    // N+2 because fft needs extra storage.
    hipfftDoubleReal *Ek;
    CUDA_CHECK(hipHostMalloc((void**)&Ek,
    			      sizeof(double) * (N/2 + 1) ));
    // device memory
    FFTArray1D d_psi = fft_alloc_1d(N);
    FFTPlan1D plan = fft_plan_create_1d(N);
    double* d_Ek;
    CUDA_CHECK(hipMalloc(&d_Ek, sizeof(double) * (N/2 + 1)) );

    std::cout << "Reading initial condition input/icond.in .." << std::endl;
    const IParams h_Iparams = read_icond("./input/icond.in");
    std::cout << "..done" << std::endl;
    std::cout << "Generating initial condition (in device) .." << std::endl;
    if (h_Iparams.FOURIER){
      set_initcond(d_psi, dk, dx, h_Iparams);
      compute_spectrum(d_psi, d_Ek);
      normalize_spectrum(d_Ek, N);
      copy_FFTArray_host_complex(psik, d_psi);
    }else{
      clean_exit_host("e2e: checking nlin works with FOURIER icond", 0);
    }
    CUDA_CHECK(hipMemcpy(Ek, d_Ek, sizeof(double) * (N/2 + 1),
			  hipMemcpyDeviceToHost));
    std::cout << "Writing intial condition to files .." << std::endl;
    write_complex_array(psik, dk, N, "inicond.out");
    std::cout << "..done" << std::endl;
    std::cout << "Writing intial energy spectrum file .." << std::endl;
    write_spectrum(Ek, N, dk, 0);
    std::cout << "..done" << std::endl;
// endsection
    std::cout << "Setup models .." << std::endl;
    setup_model(N);
    std::cout << "Testing NN conservation ..\n" ;
    hipfftDoubleComplex test = test_NN_conservation(d_psi);
    std::cout << test.x << " " <<test.y << "\n";
    std::cout << "..done \n" ;
     std::cout << "Testing calcn of nlin .." << std::endl;
    hipfftDoubleComplex* h_nlin; 
    CUDA_CHECK(hipHostMalloc(&h_nlin, sizeof(hipfftDoubleComplex) * (N/2 + 1)) );
    double* Ek_nlin;
    CUDA_CHECK(hipHostMalloc(&Ek_nlin, sizeof(double) * (N/2 + 1)) );
    copy_NLIN2host(h_nlin, Ek_nlin, d_psi);
    std::cout << "writing data .." << std::endl;
    write_complex_array(h_nlin, dk, N, "nlin.out");
    write_spectrum(Ek_nlin, N, dk, 1); 
    hipHostFree(Ek_nlin); hipHostFree(h_nlin);
    // section : clean up 
    hipHostFree(psi); hipHostFree(psik); hipHostFree(Ek); 
    fft_plan_destroy_1d(plan);
    fft_free_1d(d_psi);
    hipFree(d_Ek);
    cleanup_model();
//
    return 0;
}

