#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <complex>
#include <sys/stat.h> 
#include <hipfft/hipfft.h>      
#include "misc.h"
#include "fft_utils.h"
#include "model.h"
#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = (call);                                           \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "CUDA error %s:%d: '%s'\n", __FILE__, __LINE__, \
                    hipGetErrorString(err));                               \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    } while (0)
// ******************************************** //
// The Majda, McLaughlin, and Tabak model from J. Nonlin. Sci. Vol 6 pp 6 (1997)
struct MParams {
  double alpha;
  double beta;
  double nu;
  double Omega0 ;
  double Epsilon;
};
bool mem_allocated;
FFTPlan1D plan;
FFTArray1D NLIN;
FFTArray1D GradAlphaPsik;
double* d_Hr;
MParams h_MP;
GpuReducer red;
// Define the device constant variable only once
__constant__ MParams d_MP;
// ------------------------------------------------------
// Host function to copy parameters to device constant memory
// ------------------------------------------------------
void copy_params_to_device(const MParams& h_MP)
{
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_MP), &h_MP, sizeof(MParams)) );
}
//
void read_mparams(const char* filename){
  h_MP = {};  // zero-initialize
  h_MP.Omega0 = 1.;
  h_MP.Epsilon = 1.;
  std::ifstream file(filename);
  if (!file.is_open()) {
    std::cerr << "Error: could not open parameter file " 
	      << filename << std::endl;
    clean_exit_host("model input file not found",1);
    exit(EXIT_FAILURE);
  }
  std::string line;
  while (std::getline(file, line)) {
    std::istringstream iss(line);
    std::string key, eq;
    double value;
    
    // Expected format: key = value
    if (!(iss >> key >> eq >> value)) continue; // skip malformed lines
    
    if (key == "alpha") h_MP.alpha = value;
    else if (key == "beta") h_MP.beta = value;
    else if (key == "nu") h_MP.nu = value;
    else if (key == "Omega0") h_MP.Omega0 = value;
    else if (key == "Epsilon") h_MP.Epsilon = value;
    }    
    file.close();
}
//-------------------------------------------------
void test_model_param(){
  MParams h_MP1;
  CUDA_CHECK(hipMemcpyFromSymbol(&h_MP1, HIP_SYMBOL(d_MP), sizeof(MParams)) );
  printf("Check copied values:\n");
  std::cout << "alpha = " << h_MP1.alpha << "\n";
  std::cout << "beta  = " << h_MP1.beta  << "\n";
  std::cout << "nu    = " << (h_MP1.nu)  << "\n";
  std::cout << "Omega0    = " << h_MP1.Omega0  << "\n";
  std::cout << "Epsilon    = " << h_MP1.Epsilon  << "\n";

}
//-------------------------------
void setup_model(int N){
  // read model parameters from an input file MMT.in
  read_mparams("./input/MMT.in");
  // copy parameters to device 
  copy_params_to_device(h_MP);
  // allocate necessary device memory
  NLIN = fft_alloc_1d(N);
  GradAlphaPsik = fft_alloc_1d(N);
  plan = fft_plan_create_1d(N);
  size_t bytes = sizeof(double) * N;
  CUDA_CHECK(hipMalloc(&d_Hr, bytes));
  init_reducer(red, N);
  mem_allocated = true;
}
//
void model_free_device_memory()
{
  if(mem_allocated){
    fft_free_1d(NLIN);
    fft_free_1d(GradAlphaPsik);
    mem_allocated = false;
  }else{
    clean_exit_host("model: cannot deallocate model dev mem.", 1);
  }

}
//
void cleanup_model(){
  model_free_device_memory();
  fft_plan_destroy_1d(plan);
  free_reducer(red);
}
//----------------------------//
__device__ hipfftDoubleComplex Green(double kk){
  // G = -I * k^{alpha} - nu * k^2 
  double Omega = d_MP.Omega0 * pow(kk,d_MP.alpha);
  hipfftDoubleComplex G;
  G.x = -d_MP.nu * kk * kk;
  G.y = -Omega;
  return G;
}
//
__global__ void add_lin_kernel(hipfftDoubleComplex* Y,
			        hipfftDoubleComplex* data, int N){
    // compute the linear part and add it to the second array
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int nfreqs = N / 2 + 1;
    if (i >= nfreqs) return;
    if (i == 0) return;
   //
   double kk = (double) i;
   hipfftDoubleComplex G = Green(kk);
   hipfftDoubleComplex GY = hipCmul(G, Y[i]);
   data[i].x = data[i].x + GY.x ;
   data[i].y = data[i].y + GY.y;
}
//-----------------------
void add_lin(FFTArray1D& Y, FFTArray1D& RHS){
  // compute the linear part and add it to the second array
  int nfreqs = Y.N / 2 + 1;
  int block = 256;
  int grid = (nfreqs + block - 1) / block;
  add_lin_kernel<<<grid, block>>>(Y.d_complex, RHS.d_complex, Y.N); 
}

// ------------------------------------------------------
// Hamiltonian
// ------------------------------------------------------
__global__ void compute_Hr_kernel(double* DelAlphaby2psi, double* psi4,
				  double* d_Hamil, int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i > N) return;
  double Omega0 = d_MP.Omega0;
  double Epsilon = d_MP.Epsilon;
  double term1 = DelAlphaby2psi[i];
  double term2 = psi4[i] ;
  d_Hamil[i] = Omega0 * term1  * term1 +
    (1./2.) * Epsilon * term2 * term2 * term2 * term2;

}
//
void compute_Hr(FFTArray1D& GradAlphaby2psi, FFTArray1D& psi4){
  if(!GradAlphaby2psi.IsFourier && !psi4.IsFourier){
    int N = GradAlphaby2psi.N;
    int block = 256;
    int grid = (N + block - 1) / block;
    compute_Hr_kernel<<<grid, block>>>(GradAlphaby2psi.d_real,
				     psi4.d_real, d_Hr, N);
  }else{
    clean_exit_host("compute_Hr: should be in real space", 1);
  } 
}
//
double Hamiltonian(FFTPlan1D& plan, FFTArray1D& psik){
  int N = psik.N;
  double alpha = h_MP.alpha;
  double beta = h_MP.beta;
  copy_FFTArray(psik, NLIN); // psi(k)
  derivk(NLIN, -beta/4,  true); //|del|^{-\beta/4}psi(k)
  fft_inverse_inplace(plan, NLIN);
  normalize_fft(NLIN); //|del|^{-\beta/4}psi(r)
  
  copy_FFTArray(psik, GradAlphaPsik); // psi(k)
  derivk(GradAlphaPsik, alpha/2,  true); //|del|^{\alpha/2}psi(k)
  fft_inverse_inplace(plan, GradAlphaPsik);
  normalize_fft(GradAlphaPsik); //|del|^{\alpha/2}psi(r)
  compute_Hr(GradAlphaPsik, NLIN);
  double HH = gpu_sum(d_Hr, N, red);
  return HH;
}
//  Transform between vv and psi 
__global__ void psik_to_vv__kernel(const hipfftDoubleComplex* d_psik,
			   hipfftDoubleComplex* d_vv,
			   double time, int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int nfreqs = N / 2 + 1;
  if (i >= nfreqs) return;
  if (i == 0) return;
  //
  if (time == 0.){
    d_vv[i] = d_psik[i];
  }else{
  double kk = (double) i;
  hipfftDoubleComplex G = Green(kk);
  hipfftDoubleComplex emGt = exp_cuComplex(G, -time);
  d_vv[i] = hipCmul(emGt, d_psik[i]);
  }
}
//
__global__ void vv_to_psik__kernel(const hipfftDoubleComplex* d_vv,
			   hipfftDoubleComplex* d_psik,
			   double time, int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int nfreqs = N / 2 + 1;
  if (i >= nfreqs) return;
  if (i == 0) return;
  //
  if (time == 0.){
     d_psik[i] = d_vv[i];
  }else{
  double kk = (double) i;
  hipfftDoubleComplex G = Green(kk);
  hipfftDoubleComplex eGt = exp_cuComplex(G, time);
  d_psik[i] = hipCmul(eGt, d_vv[i]);
  }
}
void exp_inv_transform(double* vv, double* psik, double time, int N ){
  FFTArray1D Fpsik;
  FFTArray1D Fvv;
  double2FFTArray(Fpsik, psik, N);
  double2FFTArray(Fvv, vv, N);
  int nfreqs = N / 2 + 1;
  int block = 256;
  int grid = (nfreqs + block - 1) / block;
  vv_to_psik__kernel<<<grid, block>>>(Fvv.d_complex,
				      Fpsik.d_complex,
				      time, N);
}
//
void exp_transform(double* psik, double* vv, double time, int N ){
  FFTArray1D Fpsik;
  FFTArray1D Fvv;
  double2FFTArray(Fpsik, psik, N);
  double2FFTArray(Fvv, vv, N);
  int nfreqs = N / 2 + 1;
  int block = 256;
  int grid = (nfreqs + block - 1) / block;
  psik_to_vv__kernel<<<grid, block>>>(Fpsik.d_complex,
				      Fvv.d_complex,
				      time, N);
}
//---------------------
__global__ void mult_prefactor_rhsv_kernel(hipfftDoubleComplex* d_psi4,
			       hipfftDoubleComplex* vrhs,
			       double time, double dt,
			       int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int nfreqs = N / 2 + 1;
  if (i >= nfreqs) return;
  if (i == 0) return;
  //
  hipfftDoubleComplex rhs;
  double kk = (double) i;  
  if (time == 0){
    rhs.x = d_psi4[i].x ;
    rhs.y = d_psi4[i].y ;
  }else{
    hipfftDoubleComplex G = Green(kk);
    hipfftDoubleComplex emGt = exp_cuComplex(G, -time);
    rhs = hipCmul(emGt,d_psi4[i]);
  }
  vrhs[i].x =    dt * rhs.y / pow(kk, d_MP.beta)  ;
  vrhs[i].y = -  dt * rhs.x / pow(kk, d_MP.beta);    
}
//---------------------
void compute_nlin(const FFTArray1D& psik){
  copy_FFTArray(psik, NLIN); // NLIN = psi(k)
  double beta = h_MP.beta;
  double Epsilon = h_MP.Epsilon;
  derivk(NLIN, -beta/4,  true); //k^{-\beta/4}psi
  fft_inverse_inplace(plan, NLIN);
  normalize_fft(NLIN); //F^{-1}(k^{-\beta/4}psi)
  cube_FFTArray(NLIN); //( F^{-1}(k^{-\beta/4}psi) )^3 
  fft_forward_inplace(plan, NLIN); //
                          //F( (F^{-1}(|del|^{-\beta/4}psi) )^3 )
  derivk(NLIN, -beta/4, true); //
                          //(k^{-beta/4}F( (F^{-1}(|del|^{-\beta/4}psi) )^3 )
}
//---------------------
void compute_rhsv(const FFTArray1D& psik, FFTArray1D& rhs,
		  double time, double dt){
  compute_nlin(psik); // the nlin term is stored in NLIN 
  int N = psik.N;
  int nfreqs = N / 2 + 1;
  int block = 256;
  int grid = (nfreqs + block - 1) / block;
  mult_prefactor_rhsv_kernel<<<grid, block>>>(NLIN.d_complex,
				       rhs.d_complex, time, dt, N); 
}
//-------------------
hipfftDoubleComplex compute_NN_nlin(const FFTArray1D& psik){
  int N = psik.N;
  int nfreqs = N / 2 + 1;
  int block = 256;
  int grid = (nfreqs + block - 1) / block;
  mult_Astar_B<<<grid, block>>>(psik.d_complex,
				NLIN.d_complex, N);
  GpuComplexReducer ws;
  init_Complex_reducer(ws, N);
  hipfftDoubleComplex sum = gpu_Complex_sum(NLIN.d_complex,
					       N, ws);
  free_Complex_reducer(ws);
  return sum;
}
//-----------------
hipfftDoubleComplex test_NN_conservation(FFTArray1D& psik){
  if (NLIN.d_complex == nullptr)
    { printf("something is wrong \n");
      clean_exit_host("FFT1DArray NLIN not allocated", 1);
    }
  compute_nlin(psik);
  hipfftDoubleComplex Z = compute_NN_nlin(psik);
  return Z;
}
void copy_NLIN2host(hipfftDoubleComplex* h_nlin, double* h_nlink, 
		    const FFTArray1D& d_psi)
{
  if (NLIN.d_complex == nullptr)
    { printf("copy_NLIN2host: something is wrong \n");
      clean_exit_host("FFT1DArray NLIN not allocated", 1);
    }
  compute_nlin(d_psi);
  copy_FFTArray_host_complex(h_nlin, NLIN);
  int N = d_psi.N;
  double* d_nlink;
  CUDA_CHECK(hipMalloc(&d_nlink, sizeof(double) * (N/2 + 1)) );
  compute_spectrum(NLIN, d_nlink);
  normalize_spectrum(d_nlink, N);
  CUDA_CHECK(hipMemcpy(h_nlink, d_nlink, sizeof(double) * (N/2 + 1),
                          hipMemcpyDeviceToHost));
  hipFree(d_nlink);
}
// ------------------------------------------------------
// Compute RHS 
// ------------------------------------------------------
void compute_rhs(double* d_vv, double* d_psik, double* RHS,
		 double tt, double dt, int N,
		 int stage)
{
  FFTArray1D Fvv;
  FFTArray1D Fpsik;
  FFTArray1D Frhs;
  double2FFTArray(Fvv, d_vv, N);
  double2FFTArray(Fpsik, d_psik, N);
  double2FFTArray(Frhs, RHS, N);
  compute_rhsv(Fpsik, Frhs, tt, dt); 
}
